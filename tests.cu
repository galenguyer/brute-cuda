#include <hip/hip_runtime.h>
#include <stdint.h>
#include <string.h>
#include "md5.cu"

char* hash(const char* h_str) {
    char* d_str;
    unsigned char* h_res = (unsigned char*)malloc(sizeof(unsigned char)*(32 + 1));
    unsigned char* d_res;
    hipMalloc((void**)&d_str, sizeof(char) * strlen(h_str));
    hipMalloc((void**)&d_res, sizeof(char) * 32);
    hipMemcpy(d_str, h_str, sizeof(char) * strlen(h_str), hipMemcpyHostToDevice);

    md5<<<1, 1>>>(d_str, (uint32_t)strlen(h_str), d_res);

    hipMemcpy(h_res, d_res, sizeof(unsigned char)*(32), hipMemcpyDeviceToHost);

    hipFree(d_str);
    hipFree(d_res);

    char* res = (char*)malloc(sizeof(char)*32);
    for (int i = 0; i < 16; i++) {
        sprintf(&res[i*2], "%2.2x", h_res[i]);
    }
    return res;
}

int run_test(const char* name, const char* result, const char* expected) {
    if (strcmp(expected, result) == 0) {
        printf("TEST PASSED: %s: expected %s, got %s\n", name, expected, result);
        return 1;
    } else {
        printf("TEST FAILED: %s: expected %s, got %s\n", name, expected, result);
        return 0;
    }
}


int main() {
    int passed = 0, failed = 0;
    printf("----------------------------------------------------\n");
    printf("                       TESTS                        \n");
    printf("----------------------------------------------------\n");
    run_test("md5(\"\")", hash(""), "d41d8cd98f00b204e9800998ecf8427e") ? passed++ : failed++;
    run_test("md5(\"a\")", hash("a"), "0cc175b9c0f1b6a831c399e269772661") ? passed++ : failed++;
    run_test("md5(\"abc\")", hash("abc"), "900150983cd24fb0d6963f7d28e17f72") ? passed++ : failed++;
    run_test("md5(\"message digest\")", hash("message digest"), "f96b697d7cb7938d525a2f31aaf161d0") ? passed++ : failed++;
    run_test("md5(\"abcdefghijklmnopqrstuvwxyz\")", \
        hash("abcdefghijklmnopqrstuvwxyz"), \
        "c3fcd3d76192e4007dfb496cca67e13b") ? passed++ : failed++;
    run_test("md5(\"ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz0123456789\")", \
        hash("ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz0123456789"), \
        "d174ab98d277d9f5a5611c2c9f419d9f") ? passed++ : failed++;
    run_test("md5(\"12345678901234567890123456789012345678901234567890123456789012345678901234567890\")", \
        hash("12345678901234567890123456789012345678901234567890123456789012345678901234567890"), \
        "57edf4a22be3c955ac49da2e2107b67a") ? passed++ : failed++;

    printf("Tests Passed: %i\n", passed);
    printf("Tests Failed: %i\n", failed);

    printf("----------------------------------------------------\n");
    printf("                     BENCHMARKS                     \n");
    printf("----------------------------------------------------\n");
    
    for(int i = 0; i < 100000; i++) {
        hash("aa");
    }
    puts("Ran 100000 hashes");

    return failed;
}